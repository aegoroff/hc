/*
* This is an open source non-commercial project. Dear PVS-Studio, please check it.
* PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
*/
/*!
 * \brief   The file contains GPU related code implementation
 * \author  \verbatim
            Created by: Alexander Egorov
            \endverbatim
 * \date    \verbatim
            Creation date: 2017-09-27
            \endverbatim
 * Copyright: (c) Alexander Egorov 2009-2019
 */

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "gpu.h"

void gpu_get_props(device_props_t* prop) {
    struct hipDeviceProp_t device_prop;
    int n_dev_count = 0;
    
    CUDA_SAFE_CALL(hipGetDeviceCount(&n_dev_count));

    prop->device_count = n_dev_count;
    prop->max_blocks_number = 0;
    prop->max_threads_per_block = 0;

    for(int i = 0; i < n_dev_count; i++) {
        if(hipSuccess != hipGetDeviceProperties(&device_prop, i)) {
            prop->max_blocks_number += 64;
            prop->max_threads_per_block += 128;
            return;
        }
        prop->max_blocks_number += device_prop.multiProcessorCount;
        prop->max_threads_per_block += device_prop.maxThreadsPerBlock;
    }
}

BOOL gpu_can_use_gpu() {
    int n_dev_count = 0;
    hipError_t err = hipGetDeviceCount(&n_dev_count);

    if (err != hipSuccess) {
        return FALSE;
    }

    return TRUE;
}

void gpu_cleanup(gpu_tread_ctx_t* ctx) {
    CUDA_SAFE_CALL(hipHostFree(ctx->variants_));
}

void gpu_run(gpu_tread_ctx_t* ctx, const size_t dict_len, unsigned char* variants, const size_t variants_size, void(*pfn_kernel)(gpu_tread_ctx_t* c, unsigned char* r, unsigned char* v, const size_t dl)) {
    unsigned char* dev_result = nullptr;
    unsigned char* dev_variants = nullptr;

    size_t result_size_in_bytes = GPU_ATTEMPT_SIZE * sizeof(unsigned char); // include trailing zero

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_variants), variants_size * sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpyAsync(dev_variants, variants, variants_size * sizeof(unsigned char), hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_result), result_size_in_bytes));
    CUDA_SAFE_CALL(hipMemset(dev_result, 0x0, result_size_in_bytes));

#ifdef MEASURE_CUDA
    hipEvent_t start;
    hipEvent_t finish;

    lib_printf("\nVariants memory (bytes): %lli\n", variants_size);

    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&finish));

    CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif

    pfn_kernel(ctx, dev_result, dev_variants, dict_len);

    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifdef MEASURE_CUDA
    CUDA_SAFE_CALL(hipEventRecord(finish, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(finish));

    float elapsed;

    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsed, start, finish));

    lib_printf("\nCUDA Kernel time: %3.1f ms", elapsed);

    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(finish));
#endif

    CUDA_SAFE_CALL(hipMemcpy(ctx->result_, dev_result, result_size_in_bytes, hipMemcpyDeviceToHost));

    // IMPORTANT: Do not move this validation into outer scope
    // it's strange but without this call result will be undefined
    if (ctx->result_[0]) {
        ctx->found_in_the_thread_ = TRUE;
    }

    CUDA_SAFE_CALL(hipFree(dev_result));
    CUDA_SAFE_CALL(hipFree(dev_variants));
}