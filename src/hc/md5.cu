#include "hip/hip_runtime.h"
/*
* This is an open source non-commercial project. Dear PVS-Studio, please check it.
* PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
*/
/*!
 * \brief   The file contains MD5 CUDA code implementation
 * \author  \verbatim
            Created by: Alexander Egorov
            \endverbatim
 * \date    \verbatim
            Creation date: 2017-10-19
            \endverbatim
 * Copyright: (c) Alexander Egorov 2009-2017
 */

#include <stdint.h>
#include "hip/hip_runtime.h"
#include "gpu.h"
#include "md5.h"

#define DIGESTSIZE 16

 /* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

 /* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

 /* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
 /* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }


__constant__ unsigned char k_dict[CHAR_MAX];
__constant__ unsigned char k_hash[DIGESTSIZE];

__global__ static void prmd5_kernel(unsigned char* result, unsigned char* variants, const uint32_t dict_length);
__device__ static BOOL prmd5_compare(unsigned char* password, const int length);


__host__ void md5_on_gpu_prepare(int device_ix, const unsigned char* dict, size_t dict_len, const unsigned char* hash, unsigned char** variants, size_t variants_len) {
    CUDA_SAFE_CALL(hipSetDevice(device_ix));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(k_dict), dict, dict_len * sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(k_hash), hash, DIGESTSIZE));
    CUDA_SAFE_CALL(hipHostAlloc(reinterpret_cast<void**>(variants), variants_len * sizeof(unsigned char), hipHostMallocDefault));
}

__host__ void md5_on_gpu_cleanup(gpu_tread_ctx_t* ctx) {
    CUDA_SAFE_CALL(hipHostFree(ctx->variants_));
}

__host__ void md5_run_on_gpu(gpu_tread_ctx_t* ctx, const size_t dict_len, unsigned char* variants, const size_t variants_size) {
    unsigned char* dev_result = nullptr;
    unsigned char* dev_variants = nullptr;

    size_t result_size_in_bytes = GPU_ATTEMPT_SIZE * sizeof(unsigned char); // include trailing zero

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_variants), variants_size * sizeof(unsigned char)));
    CUDA_SAFE_CALL(hipMemcpyAsync(dev_variants, variants, variants_size * sizeof(unsigned char), hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&dev_result), result_size_in_bytes));
    CUDA_SAFE_CALL(hipMemset(dev_result, 0x0, result_size_in_bytes));

#ifdef MEASURE_CUDA
    hipEvent_t start;
    hipEvent_t finish;

    lib_printf("\nVariants memory (bytes): %lli\n", variants_size);

    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&finish));

    CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif
    prmd5_kernel <<<ctx->max_gpu_blocks_number_, ctx->max_threads_per_block_>>>(dev_result, dev_variants, static_cast<uint32_t>(dict_len));

    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifdef MEASURE_CUDA
    CUDA_SAFE_CALL(hipEventRecord(finish, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(finish));

    float elapsed;

    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsed, start, finish));

    lib_printf("\nCUDA Kernel time: %3.1f ms", elapsed);

    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(finish));
#endif

    CUDA_SAFE_CALL(hipMemcpy(ctx->result_, dev_result, result_size_in_bytes, hipMemcpyDeviceToHost));

    // IMPORTANT: Do not move this validation into outer scope
    // it's strange but without this call result will be undefined
    if (ctx->result_[0]) {
        ctx->found_in_the_thread_ = TRUE;
    }

    CUDA_SAFE_CALL(hipFree(dev_result));
    CUDA_SAFE_CALL(hipFree(dev_variants));
}

__global__ void prmd5_kernel(unsigned char* result, unsigned char* variants, const uint32_t dict_length) {
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned char* attempt = variants + ix * GPU_ATTEMPT_SIZE;

    size_t len = 0;

    while (attempt[len]) {
        ++len;
    }

    if (prmd5_compare(attempt, len)) {
        memcpy(result, attempt, len);
        return;
    }

    const size_t attempt_len = len + 1;

    for (uint32_t i = 0; i < dict_length; ++i)
    {
        attempt[len] = k_dict[i];

        if (prmd5_compare(attempt, attempt_len)) {
            memcpy(result, attempt, attempt_len);
            return;
        }
    }
}

__device__ __forceinline__ BOOL prmd5_compare(unsigned char* password, const int length) {
    // load into register
    const uint32_t ar = (unsigned)k_hash[0] | (unsigned)k_hash[1] << 8 | (unsigned)k_hash[2] << 16 | (unsigned)k_hash[3] << 24;
    const uint32_t br = (unsigned)k_hash[4] | (unsigned)k_hash[5] << 8 | (unsigned)k_hash[6] << 16 | (unsigned)k_hash[7] << 24;
    const uint32_t cr = (unsigned)k_hash[8] | (unsigned)k_hash[9] << 8 | (unsigned)k_hash[10] << 16 | (unsigned)k_hash[11] << 24;
    const uint32_t dr = (unsigned)k_hash[12] | (unsigned)k_hash[13] << 8 | (unsigned)k_hash[14] << 16 | (unsigned)k_hash[15] << 24;

    const uint32_t a0 = 0x67452301;
    const uint32_t b0 = 0xEFCDAB89;
    const uint32_t c0 = 0x98BADCFE;
    const uint32_t d0 = 0x10325476;

    uint32_t a = 0;
    uint32_t b = 0;
    uint32_t c = 0;
    uint32_t d = 0;

    uint32_t vals[14] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

    int i = 0;
    for (i = 0; i < length; i++) {
        vals[i / 4] |= password[i] << ((i % 4) * 8);
    }

    vals[i / 4] |= 0x80 << ((i % 4) * 8);

    const uint32_t bitlen = length * 8;

#define in0  (vals[0])//x
#define in1  (vals[1])//y
#define in2  (vals[2])//z
#define in3  (vals[3])
#define in4  (vals[4])
#define in5  (vals[5])
#define in6  (vals[6])
#define in7  (vals[7])
#define in8  (vals[8])
#define in9  (vals[9])
#define in10 (vals[10])
#define in11 (vals[11])
#define in12 (vals[12])
#define in13 (vals[13])
#define in14 (bitlen) //w = bit length
#define in15 (0)

    //Initialize hash value for this chunk:
    a = a0;
    b = b0;
    c = c0;
    d = d0;

    /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
    FF(a, b, c, d, in0, S11, 3614090360); /* 1 */
    FF(d, a, b, c, in1, S12, 3905402710); /* 2 */
    FF(c, d, a, b, in2, S13, 606105819); /* 3 */
    FF(b, c, d, a, in3, S14, 3250441966); /* 4 */
    FF(a, b, c, d, in4, S11, 4118548399); /* 5 */
    FF(d, a, b, c, in5, S12, 1200080426); /* 6 */
    FF(c, d, a, b, in6, S13, 2821735955); /* 7 */
    FF(b, c, d, a, in7, S14, 4249261313); /* 8 */
    FF(a, b, c, d, in8, S11, 1770035416); /* 9 */
    FF(d, a, b, c, in9, S12, 2336552879); /* 10 */
    FF(c, d, a, b, in10, S13, 4294925233); /* 11 */
    FF(b, c, d, a, in11, S14, 2304563134); /* 12 */
    FF(a, b, c, d, in12, S11, 1804603682); /* 13 */
    FF(d, a, b, c, in13, S12, 4254626195); /* 14 */
    FF(c, d, a, b, in14, S13, 2792965006); /* 15 */
    FF(b, c, d, a, in15, S14, 1236535329); /* 16 */

                                           /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
    GG(a, b, c, d, in1, S21, 4129170786); /* 17 */
    GG(d, a, b, c, in6, S22, 3225465664); /* 18 */
    GG(c, d, a, b, in11, S23, 643717713); /* 19 */
    GG(b, c, d, a, in0, S24, 3921069994); /* 20 */
    GG(a, b, c, d, in5, S21, 3593408605); /* 21 */
    GG(d, a, b, c, in10, S22, 38016083); /* 22 */
    GG(c, d, a, b, in15, S23, 3634488961); /* 23 */
    GG(b, c, d, a, in4, S24, 3889429448); /* 24 */
    GG(a, b, c, d, in9, S21, 568446438); /* 25 */
    GG(d, a, b, c, in14, S22, 3275163606); /* 26 */
    GG(c, d, a, b, in3, S23, 4107603335); /* 27 */
    GG(b, c, d, a, in8, S24, 1163531501); /* 28 */
    GG(a, b, c, d, in13, S21, 2850285829); /* 29 */
    GG(d, a, b, c, in2, S22, 4243563512); /* 30 */
    GG(c, d, a, b, in7, S23, 1735328473); /* 31 */
    GG(b, c, d, a, in12, S24, 2368359562); /* 32 */

                                           /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
    HH(a, b, c, d, in5, S31, 4294588738); /* 33 */
    HH(d, a, b, c, in8, S32, 2272392833); /* 34 */
    HH(c, d, a, b, in11, S33, 1839030562); /* 35 */
    HH(b, c, d, a, in14, S34, 4259657740); /* 36 */
    HH(a, b, c, d, in1, S31, 2763975236); /* 37 */
    HH(d, a, b, c, in4, S32, 1272893353); /* 38 */
    HH(c, d, a, b, in7, S33, 4139469664); /* 39 */
    HH(b, c, d, a, in10, S34, 3200236656); /* 40 */
    HH(a, b, c, d, in13, S31, 681279174); /* 41 */
    HH(d, a, b, c, in0, S32, 3936430074); /* 42 */
    HH(c, d, a, b, in3, S33, 3572445317); /* 43 */
    HH(b, c, d, a, in6, S34, 76029189); /* 44 */
    HH(a, b, c, d, in9, S31, 3654602809); /* 45 */
    HH(d, a, b, c, in12, S32, 3873151461); /* 46 */
    HH(c, d, a, b, in15, S33, 530742520); /* 47 */
    HH(b, c, d, a, in2, S34, 3299628645); /* 48 */

                                          /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
    II(a, b, c, d, in0, S41, 4096336452); /* 49 */
    II(d, a, b, c, in7, S42, 1126891415); /* 50 */
    II(c, d, a, b, in14, S43, 2878612391); /* 51 */
    II(b, c, d, a, in5, S44, 4237533241); /* 52 */
    II(a, b, c, d, in12, S41, 1700485571); /* 53 */
    II(d, a, b, c, in3, S42, 2399980690); /* 54 */
    II(c, d, a, b, in10, S43, 4293915773); /* 55 */
    II(b, c, d, a, in1, S44, 2240044497); /* 56 */
    II(a, b, c, d, in8, S41, 1873313359); /* 57 */
    II(d, a, b, c, in15, S42, 4264355552); /* 58 */
    II(c, d, a, b, in6, S43, 2734768916); /* 59 */
    II(b, c, d, a, in13, S44, 1309151649); /* 60 */
    II(a, b, c, d, in4, S41, 4149444226); /* 61 */
    II(d, a, b, c, in11, S42, 3174756917); /* 62 */
    II(c, d, a, b, in2, S43, 718787259); /* 63 */
    II(b, c, d, a, in9, S44, 3951481745); /* 64 */

    a += a0;
    b += b0;
    c += c0;
    d += d0;

    return a == ar &&
            b == br &&
            c == cr &&
            d == dr;
}
