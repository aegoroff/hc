#include "hip/hip_runtime.h"
/*
* This is an open source non-commercial project. Dear PVS-Studio, please check it.
* PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
*/
/*!
 * \brief   The file contains SHA-1 CUDA code implementation
 * \author  \verbatim
            Created by: Alexander Egorov
            \endverbatim
 * \date    \verbatim
            Creation date: 2017-09-27
            \endverbatim
 * Copyright: (c) Alexander Egorov 2009-2017
 */

#include "sha1.h"
#include "hip/hip_runtime.h"

#define DIGESTSIZE 20

extern __global__ void sha1_kernel(unsigned char* result, unsigned char* hash, const int attempt_length, const char* alphabet, const size_t abc_length);


void sha1_run_on_gpu(tread_ctx_t* ctx, const char* dict, const char* hash) {
    unsigned char* dev_result = NULL;
    char* dev_dict = NULL;
    unsigned char* dev_hash;
    size_t dict_length = strlen(dict);

    hipMalloc((void**)&dev_result, ctx->pass_length_);
    hipMalloc((void**)&dev_hash, DIGESTSIZE);
    hipMalloc((void**)&dev_dict, dict_length + 1);
    hipMemset(dev_result, 0x0, ctx->pass_length_);

    hipMemcpy(dev_hash, hash, DIGESTSIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_dict, dict, dict_length + 1, hipMemcpyHostToDevice);

    sha1_kernel <<<dict_length * dict_length, dict_length>>>(dev_result, dev_hash, ctx->pass_length_, dev_dict, dict_length);

    hipDeviceSynchronize();

    hipMemcpy(ctx->pass_, dev_result, ctx->pass_length_, hipMemcpyDeviceToHost);

    hipFree(dev_result);
    hipFree(dev_hash);
    hipFree(dev_dict);
}
